#include <stdlib.h>
#include <hip/hip_complex.h>

#include "header.h"
//#include "linear-algebra.h"
#include "complexInverse.cuh"

#ifdef DOUBLE
extern __device__ __constant__ hipDoubleComplex poles[N_RA];
extern __device__ __constant__ hipDoubleComplex res[N_RA];
#else
extern __device__ __constant__ hipFloatComplex poles[N_RA];
extern __device__ __constant__ hipFloatComplex res[N_RA];
#endif

////////////////////////////////////////////////////////////////////////

__device__
void phiAc (const double * A, const double c, double * phiA) {
	
  #ifdef DOUBLE
	hipDoubleComplex invA[NN * NN];
  #else
  	hipFloatComplex invA[NN * NN];
  #endif
	
	#pragma unroll
	for (int i = 0; i < NN * NN; ++i) {
		phiA[i] = 0.0;
	}
	
	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// takes care of (A * c - poles(q) * I)^-1
		//minv_cramer (A, c, q, invA);
		
		// compute transpose and multiply with constant
		for (int i = 0; i < NN; ++i) {
			for (int j = 0; j < NN; ++j) {
				// A - theta * I
        #ifdef DOUBLE
				if (i == j) {
					invA[i + j*NN] = hipCsub(make_hipDoubleComplex(c * A[i + j*NN], 0.0), poles[q]);
				} else {
					invA[i + j*NN] = make_hipDoubleComplex(c * A[i + j*NN], 0.0);
				}
        #else
				if (i == j) {
					invA[i + j*NN] = hipCsubf(make_hipFloatComplex(c * A[i + j*NN], 0.0), poles[q]);
				} else {
					invA[i + j*NN] = make_hipFloatComplex(c * A[i + j*NN], 0.0);
				}
        #endif
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		getComplexInverse (invA);
		
		#pragma unroll
		for (int i = 0; i < NN * NN; ++i) {
      //phiA[i] += 2.0 * creal((res[q] / poles[q]) * invA[i]);
      #ifdef DOUBLE
			phiA[i] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], poles[q]), invA[i] ) );
      #else
      		phiA[i] += 2.0 * hipCrealf( hipCmulf( hipCdivf(res[q], poles[q]), invA[i] ) );
      #endif
		}
		
	}

}