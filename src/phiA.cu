#include <stdlib.h>
#include <hip/hip_complex.h>

#include "header.h"
//#include "linear-algebra.h"
#include "complexInverse.cuh"

extern __device__ __constant__ hipDoubleComplex poles[N_RA];
extern __device__ __constant__ hipDoubleComplex res[N_RA];

////////////////////////////////////////////////////////////////////////

__device__
void phiAc (const double * A, const double c, double * phiA) {
	
	hipDoubleComplex invA[NN * NN];
	
	#pragma unroll
	for (int i = 0; i < NN * NN; ++i) {
		phiA[i] = 0.0;
	}
	
	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// takes care of (A * c - poles(q) * I)^-1
		//minv_cramer (A, c, q, invA);
		
		// compute transpose and multiply with constant
		for (int i = 0; i < NN; ++i) {
			for (int j = 0; j < NN; ++j) {
				// A - theta * I
				if (i == j) {
					invA[i + j*NN] = hipCsub(make_hipDoubleComplex(c * A[i + j*NN], 0.0), poles[q]);
				} else {
					invA[i + j*NN] = make_hipDoubleComplex(c * A[i + j*NN], 0.0);
				}
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		getComplexInverse (invA);
		
		#pragma unroll
		for (int i = 0; i < NN * NN; ++i) {
      //phiA[i] += 2.0 * creal((res[q] / poles[q]) * invA[i]);
			phiA[i] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], poles[q]), invA[i] ) );
		}
		
	}

}