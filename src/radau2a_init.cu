
#include <hip/hip_runtime.h>
/* radau2a_init.cu
*  Implementation of the necessary initialization for the Radau2A solver
 * \file radau2a_init.cu
 *
 * \author Nicholas Curtis
 * \date 03/09/2015
 *
 */

 void initialize_solver() {
 }

 const char* solver_name() {
 	const char* name = "radau2a-int-gpu";
 	return name;
 }

 void cleanup_solver() {
 	//nothing to do
 }