#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include "header.h"
#include <hip/hip_complex.h>

///////////////////////////////////////////////////////////

__device__
int getComplexMax (const int n, const hipDoubleComplex *complexArr) {
	
	int maxInd = 0;
	if (n == 1)
		return maxInd;
	
	double maxVal = hipCabs(complexArr[0]);
	for (int i = 1; i < n; ++i) {
		if (hipCabs(complexArr[i]) > maxVal) {
			maxInd = i;
			maxVal = hipCabs(complexArr[i]);
		}
	}
	
	return maxInd;
}

///////////////////////////////////////////////////////////

__device__
void scaleComplex (const int n, const hipDoubleComplex val, hipDoubleComplex* arrX) {
	
	for (int i = 0; i < n; ++i) {
		arrX[i] = hipCmul(arrX[i], val);
	}
	
}

///////////////////////////////////////////////////////////

__device__
void swapComplex (const int n, hipDoubleComplex* arrX, const int incX, hipDoubleComplex* arrY, const int incY) {
	
	int ix = 0;
	int iy = 0;
	
	for (int i = 0; i < n; ++i) {
		hipDoubleComplex temp = arrX[ix];
		arrX[ix] = arrY[iy];
		arrY[iy] = temp;
		ix += incX;
		iy += incY;
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGERU (const int n, const hipDoubleComplex alpha, const hipDoubleComplex* arrX,
									const hipDoubleComplex* arrY, const int incY, hipDoubleComplex* A, const int lda) {
	
	for (int j = 0; j < n; ++j) {
		//if (arrY[j * incY] != 0.0) {    
    if (hipCabs(arrY[j * incY]) > 0.0) {
      
			hipDoubleComplex temp = hipCmul(alpha, arrY[j * incY]);
      
			for (int i = 0; i < n; ++i) {
				//A[i + (lda * j)] += hipCmul(arrX[i], temp);
        A[i + (lda * j)] = hipCfma(arrX[i], temp, A[i + (lda * j)]);
			}
      
		}    
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexLU (const int n, hipDoubleComplex* A, int* indPivot) {
	
	//int info = 0;
	hipDoubleComplex alpha = {-1.0, 0};
	
	for (int j = 0; j < n; ++j) {
		
		// find pivot and test for singularity
		
		int jp = j + getComplexMax (n - j, &A[j + (n * j)]);
		indPivot[j] = jp;
		
		//if (A[jp + (n * j)] != 0.0) {
    if (hipCabs(A[jp + (n * j)]) > 0.0) {
			
			// apply interchange to columns 1:n-1
			if (jp != j)
				swapComplex (n, &A[j], n, &A[jp], n);
			
			// compute elements j+1:m-1 of the jth column
			
			if (j < n - 1)
				scaleComplex (n - j - 1, hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[j + (n * j)]), &A[j + 1 + (n * j)]);
			
		} //else if (info == 0) {
			//info = j + 1;
		//}
		
		// update trailing submatrix
		if (j < n - 1)
			complexGERU (n - j - 1, alpha, &A[j + 1 + (n * j)], &A[j + n * (j + 1)], n, &A[j + 1 + n * (j + 1)], n);
		
	}
	
	//return info;
}

///////////////////////////////////////////////////////////

__device__
void multiplyComplexUpperMV (const int n, hipDoubleComplex* x, const int lda, const hipDoubleComplex* A) {
	
	for (int j = 0; j < n; ++j) {
		//if (x[j] != 0.0) {
    if (hipCabs(x[j]) > 0.0) {
      
			hipDoubleComplex temp = x[j];
			for (int i = 0; i < j; ++i) {
				//x[i] += temp * A[i + (lda * j)];
        x[i] = hipCfma(temp, A[i + (lda * j)], x[i]);
			}
			//x[j] *= A[j + (lda * j)];
      x[j] = hipCmul(x[j], A[j + (lda * j)]);
		}
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGEMV (const int m, const int n, const hipDoubleComplex alpha, const hipDoubleComplex* A, 
									const hipDoubleComplex* arrX, hipDoubleComplex* arrY) {
	
	// first: y = beta*y
	// beta = 1, so nothing
	
	// second: y = alpha*A*x + y
	
	for (int j = 0; j < n - 1; ++j) {

    if (hipCabs(arrX[j]) > 0.0) {
			hipDoubleComplex temp = hipCmul(alpha, arrX[j]);
      
			for (int i = 0; i < m; ++i) {
				//arrY[i] += temp * A[i + (m * j)];
        arrY[i] = hipCfma(temp, A[i + (m * j)], arrY[i]);
			}
		}
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexInverseLU (const int n, hipDoubleComplex* A, const int* indPivot, hipDoubleComplex* work) {
	
	//int info = 0;
	
	// form inv(U)
	for (int j = 0; j < n; ++j) {
		A[j + (n * j)] = hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[j + (n * j)]);
		hipDoubleComplex Ajj = hipCmul(make_hipDoubleComplex(-1.0, 0.0), A[j + (n * j)]);
		
		// compute elements 0:j-1 of jth column
		multiplyComplexUpperMV (j, &A[n * j], n, A);
		
		// scale
		scaleComplex (j, Ajj, &A[n * j]);
	}
	
	// solve equation inv(A)*L = inv(U) for inv(A)
	
	for (int j = n - 1; j >= 0; --j) {
		
		// copy current column of L to work and replace with 0.0s
		for (int i = j + 1; i < n; ++i) {
			work[i] = A[i + (n * j)];
			A[i + (n * j)] = make_hipDoubleComplex(0.0, 0.0);
		}
		
		// compute current column of inv(A)
		if (j < n - 1)
			complexGEMV (n, n - j, make_hipDoubleComplex(-1.0, 0.0), &A[n * (j + 1)], &work[j + 1], &A[n * j]);
		
	}
	
	// apply column interchanges
	
	for (int j = n - 2; j >= 0; --j) {
    
		if (indPivot[j] != j)
			swapComplex (n, &A[n * j], 1, &A[n * indPivot[j]], 1);
	}
	
	//return info;
}

///////////////////////////////////////////////////////////

__device__
void getComplexInverse (hipDoubleComplex* A) {
	
	// pivot indices
	//int* ipiv = (int*) calloc (n, sizeof(int));
  int ipiv[NN];
	
	// output flag
	//int info = 0;
	
	// first get LU factorization
	getComplexLU (NN, A, ipiv);
	
	// check for successful exit
  /*
	if (info != 0) {
		printf ("getComplexLU failure, info = %d.\n", info);
		exit (1);
	}
  */
	
	// work array
	//hipDoubleComplex* work = (double complex*) calloc (n, sizeof(double complex));
  hipDoubleComplex work[NN];
	
	// now get inverse
	getComplexInverseLU (NN, A, ipiv, work);
	
	//free (work);
	//free (ipiv);
	
	// check for successful exit
  /*
	if (info != 0) {
		printf ("getComplexInverseLU failure, info = %d.\n", info);
		exit (1);
	}
  */
	
}