#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include "header.h"
#include "solver_props.h"
#include <hip/hip_complex.h>

///////////////////////////////////////////////////////////

__device__
int getComplexMax (const int n, const hipDoubleComplex *complexArr) {
	
	int maxInd = 0;
	if (n == 1)
		return maxInd;
	
	double maxVal = hipCabs(complexArr[0]);
	for (int i = 1; i < n; ++i) {
		if (hipCabs(complexArr[i]) > maxVal) {
			maxInd = i;
			maxVal = hipCabs(complexArr[i]);
		}
	}
	
	return maxInd;
}

///////////////////////////////////////////////////////////

__device__
void scaleComplex (const int n, const hipDoubleComplex val, hipDoubleComplex* arrX) {
	
	for (int i = 0; i < n; ++i) {
		arrX[i] = hipCmul(arrX[i], val);
	}
	
}

///////////////////////////////////////////////////////////

__device__
void swapComplex (const int n, hipDoubleComplex* arrX, const int incX, hipDoubleComplex* arrY, const int incY) {
	
	int ix = 0;
	int iy = 0;
	
	for (int i = 0; i < n; ++i) {
		hipDoubleComplex temp = arrX[ix];
		arrX[ix] = arrY[iy];
		arrY[iy] = temp;
		ix += incX;
		iy += incY;
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGERU (const int n, const hipDoubleComplex alpha, const hipDoubleComplex* arrX,
									const hipDoubleComplex* arrY, const int incY, hipDoubleComplex* A, const int lda) {
	
	for (int j = 0; j < n; ++j) {
		//if (arrY[j * incY] != 0.0) {    
    if (hipCabs(arrY[j * incY]) > 0.0) {
      
			hipDoubleComplex temp = hipCmul(alpha, arrY[j * incY]);
      
			for (int i = 0; i < n; ++i) {
				//A[i + (lda * j)] += hipCmul(arrX[i], temp);
        A[i + (lda * j)] = hipCfma(arrX[i], temp, A[i + (lda * j)]);
			}
      
		}    
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexLU (hipDoubleComplex* A, int* indPivot) {
	
	//int info = 0;
	register hipDoubleComplex alpha = make_hipDoubleComplex(-1.0, 0.0);
	
	#pragma unroll
	for (int j = 0; j < NN; ++j) {
		
		// find pivot and test for singularity
		
		int jp = j + getComplexMax (NN - j, &A[j + (NN * j)]);
		indPivot[j] = jp;
		
		//if (A[jp + (n * j)] != 0.0) {
    	if (hipCabs(A[jp + (NN * j)]) > 0.0) {
			
			// apply interchange to columns 1:n-1
			if (jp != j)
				swapComplex (NN, &A[j], NN, &A[jp], NN);
			
			// compute elements j+1:m-1 of the jth column
			
			if (j < NN - 1)
				scaleComplex (NN - j - 1, hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[j + (NN * j)]), &A[j + 1 + (NN * j)]);
			
		} //else if (info == 0) {
			//info = j + 1;
		//}
		
		// update trailing submatrix
		if (j < NN - 1)
			complexGERU (NN - j - 1, alpha, &A[j + 1 + (NN * j)], &A[j + NN * (j + 1)], NN, &A[j + 1 + NN * (j + 1)], NN);
		
	}
	
	//return info;
}

///////////////////////////////////////////////////////////

__device__
void multiplyComplexUpperMV (const int n, hipDoubleComplex* x, const int lda, const hipDoubleComplex* A) {
	
	for (int j = 0; j < n; ++j) {
		//if (x[j] != 0.0) {
    if (hipCabs(x[j]) > 0.0) {
      
			hipDoubleComplex temp = x[j];
			for (int i = 0; i < j; ++i) {
				//x[i] += temp * A[i + (lda * j)];
       			x[i] = hipCfma(temp, A[i + (lda * j)], x[i]);
			}
			//x[j] *= A[j + (lda * j)];
      		x[j] = hipCmul(x[j], A[j + (lda * j)]);
		}
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGEMV (const int m, const int n, const hipDoubleComplex alpha, const hipDoubleComplex* A, 
									const hipDoubleComplex* arrX, hipDoubleComplex* arrY) {
	
	// first: y = beta*y
	// beta = 1, so nothing
	
	// second: y = alpha*A*x + y
	
	for (int j = 0; j < n - 1; ++j) {

    if (hipCabs(arrX[j]) > 0.0) {
			hipDoubleComplex temp = hipCmul(alpha, arrX[j]);
      
			for (int i = 0; i < m; ++i) {
				//arrY[i] += temp * A[i + (m * j)];
        arrY[i] = hipCfma(temp, A[i + (NN * j)], arrY[i]);
			}
		}
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexInverseLU (const int n, hipDoubleComplex* A, const int* indPivot, hipDoubleComplex* work) {
	
	//int info = 0;
	
	// form inv(U)
	for (int j = 0; j < n; ++j) {
		A[j + (NN * j)] = hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[j + (NN * j)]);
		hipDoubleComplex Ajj = hipCmul(make_hipDoubleComplex(-1.0, 0.0), A[j + (NN * j)]);
		
		// compute elements 0:j-1 of jth column
		multiplyComplexUpperMV (j, &A[NN * j], NN, A);
		
		// scale
		scaleComplex (j, Ajj, &A[NN * j]);
	}
	
	// solve equation inv(A)*L = inv(U) for inv(A)
	
	for (int j = n - 1; j >= 0; --j) {
		
		// copy current column of L to work and replace with 0.0s
		for (int i = j + 1; i < n; ++i) {
			work[i] = A[i + (NN * j)];
			A[i + (NN * j)] = make_hipDoubleComplex(0.0, 0.0);
		}
		
		// compute current column of inv(A)
		if (j < n - 1)
			complexGEMV (n, n - j, make_hipDoubleComplex(-1.0, 0.0), &A[NN * (j + 1)], &work[j + 1], &A[NN * j]);
		
	}
	
	// apply column interchanges
	
	for (int j = n - 2; j >= 0; --j) {
    
		if (indPivot[j] != j)
			swapComplex (n, &A[NN * j], 1, &A[NN * indPivot[j]], 1);
	}
	
	//return info;
}

///////////////////////////////////////////////////////////

__device__
void getComplexInverse (hipDoubleComplex* A) {
	
	// pivot indices
	//int* ipiv = (int*) calloc (n, sizeof(int));
  	int ipiv[NN];
	
	// output flag
	//int info = 0;
	
	// first get LU factorization
	getComplexLU (A, ipiv);
	
	// check for successful exit
  /*
	if (info != 0) {
		printf ("getComplexLU failure, info = %d.\n", info);
		exit (1);
	}
  */
	
	// work array
	//hipDoubleComplex* work = (double complex*) calloc (n, sizeof(double complex));
  	hipDoubleComplex work[NN];
	
	// now get inverse
	getComplexInverseLU (NN, A, ipiv, work);
	
	//free (work);
	//free (ipiv);
	
	// check for successful exit
  /*
	if (info != 0) {
		printf ("getComplexInverseLU failure, info = %d.\n", info);
		exit (1);
	}
  */
	
}