#include <stdlib.h>
#include "header.h"
#include "solver_props.h"
//#include "linear-algebra.h"
#include "complexInverse.cuh"

#ifdef DOUBLE
extern __device__ __constant__ hipDoubleComplex poles[N_RA];
extern __device__ __constant__ hipDoubleComplex res[N_RA];
#else
extern __device__ __constant__ hipFloatComplex poles[N_RA];
extern __device__ __constant__ hipFloatComplex res[N_RA];
#endif


__device__
void phi2Ac_variable(const int m, const Real* A, const Real c, Real* phiA) {
	
#ifdef DOUBLE
	hipDoubleComplex invA[STRIDE * STRIDE];
#else
  	hipFloatComplex invA[STRIDE * STRIDE];
#endif
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[i + j*STRIDE] = ZERO;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
			#ifdef DOUBLE
				if (i == j) {
					invA[i + j*STRIDE] = hipCsub(make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0);
				}
			#else
				if (i == j) {
					invA[i + j*STRIDE] = hipCsubf(make_hipFloatComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipFloatComplex(c * A[i + j*STRIDE], 0.0);
				}
			#endif
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		//getInverseComplex (NN, invA);
		getComplexInverseHessenberg (m, invA);
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
			#ifdef DOUBLE
				phiA[i + j*STRIDE] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], hipCmul(poles[q], poles[q])), invA[i + j*STRIDE]) );
			#else
				phiA[i + j*STRIDE] += 2.0 * hipCrealf( hipCmulf( hipCdivf(res[q], hipCmulf(poles[q], poles[q])), invA[i + j*STRIDE]) );
			#endif
			}
		}
	}
}

__device__
void phiAc_variable(const int m, const Real* A, const Real c, Real* phiA) {
	
#ifdef DOUBLE
	hipDoubleComplex invA[STRIDE * STRIDE];
#else
  	hipFloatComplex invA[STRIDE * STRIDE];
#endif
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[i + j*STRIDE] = ZERO;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
			#ifdef DOUBLE
				if (i == j) {
					invA[i + j*STRIDE] = hipCsub(make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0);
				}
			#else
				if (i == j) {
					invA[i + j*STRIDE] = hipCsubf(make_hipFloatComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipFloatComplex(c * A[i + j*STRIDE], 0.0);
				}
			#endif
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		//getInverseComplex (NN, invA);
		getComplexInverseHessenberg (m, invA);
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
			#ifdef DOUBLE
				phiA[i + j*STRIDE] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], poles[q]), invA[i + j*STRIDE]) );
			#else
				phiA[i + j*STRIDE] += 2.0 * hipCrealf( hipCmulf( hipCdivf(res[q], poles[q]), invA[i + j*STRIDE]) );
			#endif
			}
		}
	}
}

__device__
void expAc_variable(const int m, const Real* A, const Real c, Real* phiA) {
	
#ifdef DOUBLE
	hipDoubleComplex invA[STRIDE * STRIDE];
#else
  	hipFloatComplex invA[STRIDE * STRIDE];
#endif
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[i + j*STRIDE] = ZERO;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
			#ifdef DOUBLE
				if (i == j) {
					invA[i + j*STRIDE] = hipCsub(make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0);
				}
			#else
				if (i == j) {
					invA[i + j*STRIDE] = hipCsubf(make_hipFloatComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipFloatComplex(c * A[i + j*STRIDE], 0.0);
				}
			#endif
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		//getInverseComplex (NN, invA);
		getComplexInverseHessenberg (m, invA);
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
			#ifdef DOUBLE
				phiA[i + j*STRIDE] += 2.0 * hipCreal( hipCmul( res[q], invA[i + j*STRIDE]) );
			#else
				phiA[i + j*STRIDE] += 2.0 * hipCrealf( hipCmulf( res[q], invA[i + j*STRIDE]) )
			#endif
			}
		}
	}
}