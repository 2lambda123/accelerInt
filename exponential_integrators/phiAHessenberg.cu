#include <stdlib.h>
#include "header.cuh"
#include "solver_options.h"
#include "solver_props.cuh"
//#include "linear-algebra.h"
#include "complexInverse.cuh"

extern __device__ __constant__ hipDoubleComplex poles[N_RA];
extern __device__ __constant__ hipDoubleComplex res[N_RA];

__device__
void phi2Ac_variable(const int m, const double* A, const double c, double* phiA) {
	
	hipDoubleComplex invA[STRIDE * STRIDE];
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[i + j*STRIDE] = 0.0;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
				if (i == j) {
					invA[i + j*STRIDE] = hipCsub(make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0);
				}
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		//getInverseComplex (NN, invA);
		getComplexInverseHessenberg (m, invA);
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
				phiA[i + j*STRIDE] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], hipCmul(poles[q], poles[q])), invA[i + j*STRIDE]) );
			}
		}
	}
}

__device__
void phiAc_variable(const int m, const double* A, const double c, double* phiA) {

	hipDoubleComplex invA[STRIDE * STRIDE];
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[i + j*STRIDE] = 0.0;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
				if (i == j) {
					invA[i + j*STRIDE] = hipCsub(make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0);
				}
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		//getInverseComplex (NN, invA);
		getComplexInverseHessenberg (m, invA);
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
				phiA[i + j*STRIDE] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], poles[q]), invA[i + j*STRIDE]) );
			}
		}
	}
}

__device__
void expAc_variable(const int m, const double* A, const double c, double* phiA) {

	hipDoubleComplex invA[STRIDE * STRIDE];
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[i + j*STRIDE] = 0.0;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
				if (i == j) {
					invA[i + j*STRIDE] = hipCsub(make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0), poles[q]);
				} else {
					invA[i + j*STRIDE] = make_hipDoubleComplex(c * A[i + j*STRIDE], 0.0);
				}
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		//getInverseComplex (NN, invA);
		getComplexInverseHessenberg (m, invA);
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
				phiA[i + j*STRIDE] += 2.0 * hipCreal( hipCmul( res[q], invA[i + j*STRIDE]) );
			}
		}
	}
}