#include <stdlib.h>
#include "header.cuh"
#include "solver_options.cuh"
#include "solver_props.cuh"
//#include "linear-algebra.h"
#include "complexInverse.cuh"

extern __device__ __constant__ hipDoubleComplex poles[N_RA];
extern __device__ __constant__ hipDoubleComplex res[N_RA];

__device__
int phi2Ac_variable(const int m, const double* __restrict__ A, const double c,
						double* __restrict__ phiA, const solver_memory* __restrict__ solver,
						hipDoubleComplex* __restrict__ work) {
	
	hipDoubleComplex * const __restrict__ invA = solver->invA;
	int * const __restrict__ ipiv = solver->ipiv;
	int info = 0;
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[INDEX(i + j*STRIDE)] = 0.0;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
				if (i == j) {
					invA[INDEX(i + j*STRIDE)] = hipCsub(make_hipDoubleComplex(c * A[INDEX(i + j*STRIDE)], 0.0), poles[q]);
				} else {
					invA[INDEX(i + j*STRIDE)] = make_hipDoubleComplex(c * A[INDEX(i + j*STRIDE)], 0.0);
				}
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		getComplexInverseHessenberg (m, invA, ipiv, &info, work);

		if (info != 0)
			return info;
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
				phiA[INDEX(i + j*STRIDE)] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], hipCmul(poles[q], poles[q])), invA[INDEX(i + j*STRIDE)]) );
			}
		}
	}
	return 0;
}

__device__
int phiAc_variable(const int m, const double* __restrict__ A, const double c,
						double* __restrict__ phiA, const solver_memory* __restrict__ solver,
						hipDoubleComplex* __restrict__ work) {
	
	hipDoubleComplex * const __restrict__ invA = solver->invA;
	int * const __restrict__ ipiv = solver->ipiv;
	int info = 0;

	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[INDEX(i + j*STRIDE)] = 0.0;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
				if (i == j) {
					invA[INDEX(i + j*STRIDE)] = hipCsub(make_hipDoubleComplex(c * A[INDEX(i + j*STRIDE)], 0.0), poles[q]);
				} else {
					invA[INDEX(i + j*STRIDE)] = make_hipDoubleComplex(c * A[INDEX(i + j*STRIDE)], 0.0);
				}
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		getComplexInverseHessenberg (m, invA, ipiv, &info, work);

		if (info != 0)
			return info;
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
				phiA[INDEX(i + j*STRIDE)] += 2.0 * hipCreal( hipCmul( hipCdiv(res[q], poles[q]), invA[INDEX(i + j*STRIDE)]) );
			}
		}
	}
	return 0;
}

__device__
int expAc_variable(const int m, const double* __restrict__ A, const double c,
						double* __restrict__ phiA, const solver_memory* __restrict__ solver,
						hipDoubleComplex* __restrict__ work) {
	
	hipDoubleComplex * const __restrict__ invA = solver->invA;
	int * const __restrict__ ipiv = solver->ipiv;
	int info = 0;
	
	#pragma unroll
	for (int i = 0; i < m; ++i) {
		#pragma unroll
		for (int j = 0; j < m; ++j) {
			phiA[INDEX(i + j*STRIDE)] = 0.0;
		}
	}

	#pragma unroll
	for (int q = 0; q < N_RA; q += 2) {
		
		// compute transpose and multiply with constant
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < m; ++j) {
				// A - theta * I
				if (i == j) {
					invA[INDEX(i + j*STRIDE)] = hipCsub(make_hipDoubleComplex(c * A[INDEX(i + j*STRIDE)], 0.0), poles[q]);
				} else {
					invA[INDEX(i + j*STRIDE)] = make_hipDoubleComplex(c * A[INDEX(i + j*STRIDE)], 0.0);
				}
			}
		}
		
		// takes care of (A * c - poles(q) * I)^-1
		getComplexInverseHessenberg (m, invA, ipiv, &info, work);

		if (info != 0)
			return info;
		
		
		#pragma unroll
		for (int i = 0; i < m; ++i) {
			#pragma unroll
			for (int j = 0; j < m; ++j) {
				phiA[INDEX(i + j*STRIDE)] += 2.0 * hipCreal( hipCmul( res[q], invA[INDEX(i + j*STRIDE)]) );
			}
		}
	}
	return 0;
}