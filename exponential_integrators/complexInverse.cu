#include "header.cuh"
#include "solver_props.cuh"
#include <hip/hip_complex.h>
///////////////////////////////////////////////////////////

__device__
int getComplexMax (const int n, const hipDoubleComplex *complexArr) {
	
	int maxInd = 0;
	if (n == 1)
		return maxInd;
	
	double maxVal = hipCabs(complexArr[0]);
	for (int i = 1; i < n; ++i) {
		if (hipCabs(complexArr[i]) > maxVal) {
			maxInd = i;
			maxVal = hipCabs(complexArr[i]);
		}
	}
	
	return maxInd;
}

///////////////////////////////////////////////////////////

__device__
void scaleComplex (const int n, const hipDoubleComplex val, hipDoubleComplex* arrX) {
	
	for (int i = 0; i < n; ++i) {
		arrX[i] = hipCmul(arrX[i], val);
	}
	
}

///////////////////////////////////////////////////////////

__device__
void swapComplex (const int n, hipDoubleComplex* arrX, const int incX, hipDoubleComplex* arrY, const int incY) {
	
	int ix = 0;
	int iy = 0;
	
	for (int i = 0; i < n; ++i) {
		hipDoubleComplex temp = arrX[ix];
		arrX[ix] = arrY[iy];
		arrY[iy] = temp;
		ix += incX;
		iy += incY;
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGERU (const int n, const hipDoubleComplex alpha, const hipDoubleComplex* arrX,
									const hipDoubleComplex* arrY, const int incY, hipDoubleComplex* A, const int lda) {
	
	for (int j = 0; j < n; ++j) {
    	if (hipCabs(arrY[j * incY]) > 0.0) {
      
			hipDoubleComplex temp = hipCmul(alpha, arrY[j * incY]);
      
			for (int i = 0; i < n; ++i) {
				A[i + (lda * j)] = hipCfma(arrX[i], temp, A[i + (lda * j)]);
			}
      
		}    
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexLU (const int n, hipDoubleComplex* A, int* indPivot) {
	
	//int info = 0;
	for (int j = 0; j < n; ++j) {
		
		// find pivot and test for singularity
		
		int jp = j + getComplexMax (n - j, &A[j + (STRIDE * j)]);
		indPivot[j] = jp;

    	if (hipCabs(A[jp + (STRIDE * j)]) > 0.0) {
			
			// apply interchange to columns 1:n-1
			if (jp != j)
				swapComplex (n, &A[j], STRIDE, &A[jp], STRIDE);
			
			// compute elements j+1:m-1 of the jth column
			
			if (j < n - 1)
				scaleComplex (n - j - 1, hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[j + (STRIDE * j)]), &A[j + 1 + (STRIDE * j)]);
			
		} //else if (info == 0) {
			//info = j + 1;
		//}
		
		// update trailing submatrix
		if (j < n - 1)
			complexGERU (n - j - 1, make_hipDoubleComplex(-1.0, 0.0), &A[j + 1 + (STRIDE * j)], &A[j + STRIDE * (j + 1)], STRIDE, &A[j + 1 + STRIDE * (j + 1)], STRIDE);
		
	}
	
	//return info;
}

///////////////////////////////////////////////////////////

__device__
void multiplyComplexUpperMV (const int n, hipDoubleComplex* x, const int lda, const hipDoubleComplex* A) {
	
	for (int j = 0; j < n; ++j) {
		//if (x[j] != 0.0) {
    if (hipCabs(x[j]) > 0.0) {
      
			hipDoubleComplex temp = x[j];
			for (int i = 0; i < j; ++i) {
				//x[i] += temp * A[i + (lda * j)];
       			x[i] = hipCfma(temp, A[i + (lda * j)], x[i]);
			}
			//x[j] *= A[j + (lda * j)];
      		x[j] = hipCmul(x[j], A[j + (lda * j)]);
		}
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGEMV (const int m, const int n, const hipDoubleComplex alpha, const hipDoubleComplex* A, 
									const hipDoubleComplex* arrX, hipDoubleComplex* arrY) {
	
	// first: y = beta*y
	// beta = 1, so nothing
	
	// second: y = alpha*A*x + y
	
	for (int j = 0; j < n - 1; ++j) {

    if (hipCabs(arrX[j]) > 0.0) {
			hipDoubleComplex temp = hipCmul(alpha, arrX[j]);
      
			for (int i = 0; i < m; ++i) {
				//arrY[i] += temp * A[i + (m * j)];
        		 arrY[i] = hipCfma(temp, A[i + (STRIDE * j)], arrY[i]);
			}
		}
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexInverseLU (const int n, hipDoubleComplex* A, const int* indPivot, hipDoubleComplex* work) {
	
	//int info = 0;
	
	// form inv(U)
	for (int j = 0; j < n; ++j) {
		A[j + (STRIDE * j)] = hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[j + (STRIDE * j)]);
		hipDoubleComplex Ajj = hipCmul(make_hipDoubleComplex(-1.0, 0.0), A[j + (STRIDE * j)]);
		
		// compute elements 0:j-1 of jth column
		multiplyComplexUpperMV (j, &A[STRIDE * j], STRIDE, A);
		
		// scale
		scaleComplex (j, Ajj, &A[STRIDE * j]);
	}
	
	// solve equation inv(A)*L = inv(U) for inv(A)
	
	for (int j = n - 1; j >= 0; --j) {
		
		// copy current column of L to work and replace with 0.0s
		for (int i = j + 1; i < n; ++i) {
			work[i] = A[i + (STRIDE * j)];
			A[i + (STRIDE * j)] = make_hipDoubleComplex(0.0, 0.0);
		}
		
		// compute current column of inv(A)
		if (j < n - 1)
			complexGEMV (n, n - j, make_hipDoubleComplex(-1.0, 0.0), &A[STRIDE * (j + 1)], &work[j + 1], &A[STRIDE * j]);
		
	}
	
	// apply column interchanges
	
	for (int j = n - 2; j >= 0; --j) {
    
		if (indPivot[j] != j)
			swapComplex (n, &A[STRIDE * j], 1, &A[STRIDE * indPivot[j]], 1);
	}
	
	//return info;
}

///////////////////////////////////////////////////////////

__device__
void getComplexInverse (int n, hipDoubleComplex* A) {
	
	// pivot indices
	//int* ipiv = (int*) calloc (n, sizeof(int));
  	int ipiv[STRIDE];
	
	// output flag
	//int info = 0;
	
	// first get LU factorization
	getComplexLU (n, A, ipiv);
	
	// check for successful exit
  /*
	if (info != 0) {
		printf ("getComplexLU failure, info = %d.\n", info);
		exit (1);
	}
  */
	
	// work array
	//hipDoubleComplex* work = (double complex*) calloc (n, sizeof(double complex));
  	hipDoubleComplex work[STRIDE];
	
	// now get inverse
	getComplexInverseLU (n, A, ipiv, work);
	
	//free (work);
	//free (ipiv);
	
	// check for successful exit
  /*
	if (info != 0) {
		printf ("getComplexInverseLU failure, info = %d.\n", info);
		exit (1);
	}
  */
	
}

//Matrix Algorithms: Volume 1: Basic Decompositions
//By G. W. Stewart
__device__
void getHessenbergLU(const int n, hipDoubleComplex* A, int* indPivot)
{
	int last_free = 0;
	for (int i = 0; i < n - 1; i ++)
	{
		if (hipCabs(A[i * STRIDE + i]) < hipCabs(A[i * STRIDE + i + 1]))
		{
			//swap rows
			swapComplex(n - last_free, &A[last_free * STRIDE + i], STRIDE, &A[last_free * STRIDE + i + 1], STRIDE);
			indPivot[i] = i + 1;
		}
		else
		{
			indPivot[i] = i;
			last_free = i;
		}
		if (hipCabs(A[i * STRIDE + i]) > 0.0)
		{
			hipDoubleComplex tau = hipCdiv(A[i * STRIDE + i + 1], A[i * STRIDE + i]);
			for (int j = i + 1; j < n; j++)
			{
				A[j * STRIDE + i + 1] = hipCsub(A[j * STRIDE + i + 1], hipCmul(tau, A[j * STRIDE + i]));
			}
			A[i * STRIDE + i + 1] = tau;
		}
	}
	//last index is not pivoted
	indPivot[n - 1] = n - 1;
}

__device__
void getComplexInverseHessenberg (const int n, hipDoubleComplex* A)
{
	// pivot indices
	int ipiv[STRIDE];
	
	// first get LU factorization
	getHessenbergLU (n, A, ipiv);

	// work array
	hipDoubleComplex work[STRIDE];
	
	// now get inverse
	getComplexInverseLU (n, A, ipiv, work);
}