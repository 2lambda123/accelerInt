#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include "header.cuh"
#include "solver_props.cuh"

///////////////////////////////////////////////////////////

__device__
int getMax (const int n, const double *Arr) {
	
	int maxInd = 0;
	if (n == 1)
		return maxInd;
	
	double maxVal = fabs(Arr[0]);
	for (int i = 1; i < n; ++i) {
		if (fabs(Arr[i]) > maxVal) {
			maxInd = i;
			maxVal = fabs(Arr[i]);
		}
	}
	
	return maxInd;
}

///////////////////////////////////////////////////////////

__device__
void scale (const int n, const double val, double* arrX) {
	
	for (int i = 0; i < n; ++i) {
		arrX[i] *= val;
	}
	
}

///////////////////////////////////////////////////////////

__device__
void swap (const int n, double* arrX, const int incX, double* arrY, const int incY) {
	
	int ix = 0;
	int iy = 0;
	
	for (int i = 0; i < n; ++i) {
		double temp = arrX[ix];
		arrX[ix] = arrY[iy];
		arrY[iy] = temp;
		ix += incX;
		iy += incY;
	}
	
}

///////////////////////////////////////////////////////////

__device__
void GERU (const int n, const double alpha, const double* arrX,
									const double* arrY, const int incY, double* A, const int lda) {
	
	for (int j = 0; j < n; ++j) {
    	if (fabs(arrY[j * incY]) > 0.0) {
      
			double temp = alpha * arrY[j * incY];
      
			for (int i = 0; i < n; ++i) {
				A[i + (lda * j)] += arrX[i] * temp;
			}
      
		}    
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getLU (double* A, int* indPivot, int* info) {
	
	#pragma unroll
	for (int j = 0; j < NSP; ++j) {
		
		// find pivot and test for singularity
		
		int jp = j + getMax (NSP - j, &A[j + (NSP * j)]);
		indPivot[j] = jp;

    	if (fabs(A[jp + (NSP * j)]) > 0.0) {
			
			// apply interchange to columns 1:n-1
			if (jp != j)
				swap(NSP, &A[j], NSP, &A[jp], NSP);
			
			// compute elements j+1:m-1 of the jth column
			
			if (j < NSP - 1)
				scale(NSP - j - 1, 1.0 / A[j + (NSP * j)], &A[j + 1 + (NSP * j)]);
			
		} else if (*info == 0) {
			*info = j;
			break;
		}
		
		// update trailing submatrix
		if (j < NSP - 1)
			GERU (NSP - j - 1, -1.0, &A[j + 1 + (NSP * j)], &A[j + NSP * (j + 1)], NSP, &A[j + 1 + NSP * (j + 1)], NSP);	
	}
}