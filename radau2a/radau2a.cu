#include "hip/hip_runtime.h"
/** 
* \file radau2a.cu
*
* \author Nicholas J. Curtis
* \date 03/16/2015
*
* A Radau2A IRK implementation for CUDA
* 
* NOTE: all matricies stored in column major format!
* 
*/

#include "header.cuh"
#include "solver_options.h"
#include "inverse.cuh"
#include "complexInverse_NSP.cuh"
#include "solver_options.h"
#include "jacob.cuh"
#include "dydt.cuh"
#include <hip/hip_complex.h>

#ifdef NEWTON_UNROLL
	#define NEWTON_UNROLLER	#pragma unroll NEWTON_UNROLL
#else
	#define NEWTON_UNROLLER #pragma unroll
#endif
#ifdef WARP_VOTING
	#define ANY(X) (__any((X)))
	#define ALL(X) (__all((X)))
#else
	#define ANY(X) ((X))
	#define ALL(X) ((X))
#endif
#define Max_no_steps (200000)
#define NewtonMaxit (8)
#define StartNewton (true)
#define Gustafsson
#define Roundoff (EPS)
#define FacMin (0.2)
#define FacMax (8)
#define FacSafe (0.9)
#define FacRej (0.1)
#define ThetaMin (0.001)
#define NewtonTol (0.03)
#define Qmin (1.0)
#define Qmax (1.2)
#define UNROLL (8)
//#define SDIRK_ERROR

__device__
void scale (const double * y0, const double* y, double * sc) {
	#pragma unroll 1
	for (int i = 0; i < NSP; ++i) {
		sc[i] = 1.0 / (ATOL + fmax(fabs(y0[i]), fabs(y[i])) * RTOL);
	}
}

__device__
void scale_init (const double * y0, double * sc) {
	#pragma unroll 1
	for (int i = 0; i < NSP; ++i) {
		sc[i] = 1.0 / (ATOL + fabs(y0[i]) * RTOL);
	}
}

__device__
void safe_memcpy(double* dest, const double* source)
{
	#pragma unroll 1
	for (int i = 0; i < NSP; i++)
	{
		dest[i] = source[i];
	}
}
__device__
void safe_memset3(double* dest1, double* dest2, double* dest3, const double val)
{
	#pragma unroll 1
	for (int i = 0; i < NSP; i++)
	{
		dest1[i] = val;
		dest2[i] = val;
		dest3[i] = val;
	}
}

__constant__ double rkA[3][3] = { {
	 1.968154772236604258683861429918299e-1,
	-6.55354258501983881085227825696087e-2,
	 2.377097434822015242040823210718965e-2
	}, {
	 3.944243147390872769974116714584975e-1,
	 2.920734116652284630205027458970589e-1,
	-4.154875212599793019818600988496743e-2
	}, {
	 3.764030627004672750500754423692808e-1,
	 5.124858261884216138388134465196080e-1,
	 1.111111111111111111111111111111111e-1
	}
};

__constant__ double rkB[3] = {
3.764030627004672750500754423692808e-1,
5.124858261884216138388134465196080e-1,
1.111111111111111111111111111111111e-1
};

__constant__ double rkC[3] = {
1.550510257216821901802715925294109e-1,
6.449489742783178098197284074705891e-1,
1.0
};

#ifdef SDIRK_ERROR
	// Classical error estimator: 
	// H* Sum (B_j-Bhat_j)*f(Z_j) = H*E(0)*f(0) + Sum E_j*Z_j
	__constant__ double rkE[4] = {
	0.02,
	-10.04880939982741556246032950764708e0*0.02,
	1.382142733160748895793662840980412e0*0.02,
	-0.3333333333333333333333333333333333e0*0.02
	};
	// H* Sum Bgam_j*f(Z_j) = H*Bgam(0)*f(0) + Sum Theta_j*Z_j
	__constant__ double rkTheta[3] = {
	-1.520677486405081647234271944611547e0 - 10.04880939982741556246032950764708e0*0.02,
	2.070455145596436382729929151810376e0 + 1.382142733160748895793662840980413e0*0.02,
	-0.3333333333333333333333333333333333e0*0.02 - 0.3744441479783868387391430179970741
	};
	// ! Sdirk error estimator
	__constant__ double rkBgam[5] = {
	0.02,
	0.3764030627004672750500754423692807-1.558078204724922382431975370686279*0.02,
	0.8914115380582557157653087040196118*0.02+.5124858261884216138388134465196077,
	-0.1637777184845662566367174924883037-0.3333333333333333333333333333333333*0.02,
	0.2748888295956773677478286035994148
	};
#else
	// Classical error estimator: 
	// H* Sum (B_j-Bhat_j)*f(Z_j) = H*E(0)*f(0) + Sum E_j*Z_j
	__constant__ double rkE[4] = {
	0.05,
	-10.04880939982741556246032950764708e0*0.05,
	1.382142733160748895793662840980412e0*0.05,
	-0.3333333333333333333333333333333333e0*0.05
	};
	// H* Sum Bgam_j*f(Z_j) = H*Bgam(0)*f(0) + Sum Theta_j*Z_j
	__constant__ double rkTheta[3] = {
	-1.520677486405081647234271944611547e0 - 10.04880939982741556246032950764708e0*0.05,
	2.070455145596436382729929151810376e0 + 1.382142733160748895793662840980413e0*0.05,
	-0.3333333333333333333333333333333333e0*0.05 - 0.3744441479783868387391430179970741e0
	};
#endif

//Local order of error estimator 
/*
!~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
!~~~> Diagonalize the RK matrix:               
! rkTinv * inv(rkA) * rkT =          
!           |  rkGamma      0           0     |
!           |      0      rkAlpha   -rkBeta   |
!           |      0      rkBeta     rkAlpha  |
!~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
*/

__constant__ double rkGamma = 3.637834252744495732208418513577775e0;
__constant__ double rkAlpha = 2.681082873627752133895790743211112e0;
__constant__ double rkBeta  = 3.050430199247410569426377624787569e0;

__constant__ double rkT[3][3] = {
{9.443876248897524148749007950641664e-2,
-1.412552950209542084279903838077973e-1,
-3.00291941051474244918611170890539e-2},
{2.502131229653333113765090675125018e-1,
2.041293522937999319959908102983381e-1,
3.829421127572619377954382335998733e-1},
{1.0e0,
1.0e0,
0.0e0}
};

__constant__ double rkTinv[3][3] = {
{4.178718591551904727346462658512057e0,
3.27682820761062387082533272429617e-1,
5.233764454994495480399309159089876e-1},
{-4.178718591551904727346462658512057e0,
-3.27682820761062387082533272429617e-1,
4.766235545005504519600690840910124e-1},
{-5.02872634945786875951247343139544e-1,
2.571926949855605429186785353601676e0,
-5.960392048282249249688219110993024e-1}
};

__constant__ double rkTinvAinv[3][3] = {
{1.520148562492775501049204957366528e+1,
1.192055789400527921212348994770778e0,
1.903956760517560343018332287285119e0},
{-9.669512977505946748632625374449567e0,
-8.724028436822336183071773193986487e0,
3.096043239482439656981667712714881e0},
{-1.409513259499574544876303981551774e+1,
5.895975725255405108079130152868952e0,
-1.441236197545344702389881889085515e-1}
};

__constant__ double rkAinvT[3][3] = {
{0.3435525649691961614912493915818282e0,
-0.4703191128473198422370558694426832e0,
0.3503786597113668965366406634269080e0},
{0.9102338692094599309122768354288852e0,
1.715425895757991796035292755937326e0,
0.4040171993145015239277111187301784e0},
{3.637834252744495732208418513577775e0,
2.681082873627752133895790743211112e0,
-3.050430199247410569426377624787569e0}
};

__constant__ double rkELO = 4;

///////////////////////////////////////////////////////////////////////////////

/*
* calculate E1 & E2 matricies and their LU Decomposition
*/
__device__ void RK_Decomp(double H, double* E1, hipDoubleComplex* E2, const double* Jac, int* ipiv1, int* ipiv2, int* info) {
	hipDoubleComplex temp = make_hipDoubleComplex(rkAlpha/H, rkBeta/H);
	#pragma unroll 1
	for (int i = 0; i < NSP; i++)
	{
		#pragma unroll 1
		for(int j = 0; j < NSP; j++)
		{
			E1[i + j * NSP] = -Jac[i + j * NSP];
			E2[i + j * NSP] = make_hipDoubleComplex(-Jac[i + j * NSP], 0);
		}
		E1[i + i * NSP] += rkGamma / H;
		E2[i + i * NSP] = hipCadd(E2[i + i * NSP], temp); 
	}
	getLU(E1, ipiv1, info);
	if (*info != 0) {
		return;
	}
	getComplexLU(E2, ipiv2, info);
}

__device__ void RK_Make_Interpolate(const double* Z1, const double* Z2, const double* Z3, double* CONT) {
	double den = (rkC[2] - rkC[1]) * (rkC[1] - rkC[0]) * (rkC[0] - rkC[2]); 
	#pragma unroll 1
	for (int i = 0; i < NSP; i++) {
		CONT[i] = ((-rkC[2] * rkC[2] * rkC[1] * Z1[i] + Z3[i] * rkC[1]* rkC[0] * rkC[0]
                    + rkC[1] * rkC[1] * rkC[2] * Z1[i] - rkC[1] * rkC[1] * rkC[0] * Z3[i] 
                    + rkC[2] * rkC[2] * rkC[0] * Z2[i] - Z2[i] * rkC[2] * rkC[0] * rkC[0])
                    /den)-Z3[i];
        CONT[NSP + i] = -( rkC[0] * rkC[0] * (Z3[i] - Z2[i]) + rkC[1] * rkC[1] * (Z1[i] - Z3[i]) 
        				 + rkC[2] * rkC[2] * (Z2[i] - Z1[i]) )/den;
        CONT[NSP + NSP + i] = ( rkC[0] * (Z3[i] - Z2[i]) + rkC[1] * (Z1[i] - Z3[i]) 
                           + rkC[2] * (Z2[i] - Z1[i]) ) / den;
	}
}

__device__ void RK_Interpolate(double H, double Hold, double* Z1, double* Z2, double* Z3, const double* CONT) {
	double r = H / Hold;
	register double x1 = 1.0 + rkC[0] * r;
	register double x2 = 1.0 + rkC[1] * r;
	register double x3 = 1.0 + rkC[2] * r;
	#pragma unroll 1
	for (int i = 0; i < NSP; i++) {
		Z1[i] = CONT[i] + x1 * (CONT[NSP + i] + x1 * CONT[NSP + NSP + i]);
		Z2[i] = CONT[i] + x2 * (CONT[NSP + i] + x2 * CONT[NSP + NSP + i]);
		Z3[i] = CONT[i] + x2 * (CONT[NSP + i] + x3 * CONT[NSP + NSP + i]);
	}
}


__device__ void WADD(const double* X, const double* Y, double* Z) {
	#pragma unroll 1
	for (int i = 0; i < NSP; i++)
	{
		Z[i] = X[i] + Y[i];
	}
}

__device__ void DAXPY3(double DA1, double DA2, double DA3, const double* DX, double* DY1, double* DY2, double* DY3) {
	#pragma unroll 1
	for (int i = 0; i < NSP; i++) {
		DY1[i] += DA1 * DX[i];
		DY2[i] += DA2 * DX[i];
		DY3[i] += DA3 * DX[i];
	}
}

/*
*Prepare the right-hand side for Newton iterations
*     R = Z - hA * F
*/
__device__ void RK_PrepareRHS(double t, double pr, double H, double* Y, double* F0, double* Z1, double* Z2, double* Z3, double* R1, double* R2, double* R3) {
	double TMP[NSP];
	double F[NSP];
	#pragma unroll 1
	for (int i = 0; i < NSP; i++) {
		R1[i] = Z1[i];
		R2[i] = Z2[i];
		R3[i] = Z3[i];
	}

	// TMP = Y + Z1
	WADD(Y, Z1, TMP);
	dydt(t + rkC[0] * H, pr, TMP, F);
	//R[:] -= -h * rkA[:][0] * F[:]
	DAXPY3(-H * rkA[0][0], -H * rkA[1][0], -H * rkA[2][0], F, R1, R2, R3);

	// TMP = Y + Z2
	WADD(Y, Z2, TMP);
	dydt(t + rkC[1] * H, pr, TMP, F);
	//R[:] -= -h * rkA[:][1] * F[:]
	DAXPY3(-H * rkA[0][1], -H * rkA[1][1], -H * rkA[2][1], F, R1, R2, R3);

	// TMP = Y + Z3
	WADD(Y, Z3, TMP);
	dydt(t + rkC[2] * H, pr, TMP, F);
	//R[:] -= -h * rkA[:][2] * F[:]
	DAXPY3(-H * rkA[0][2], -H * rkA[1][2], -H * rkA[2][2], F, R1, R2, R3);
}

__device__ void dlaswp(double* A, int* ipiv) {
	#pragma unroll 1
	for (int i = 0; i < NSP; i++) {
		int ip = ipiv[i];
		if (ip != i) {
			double temp = A[i];
			A[i] = A[ip];
			A[ip] = temp;
		}
	}	
}

//diag == 'n' -> nounit = true
//upper == 'u' -> upper = true
__device__ void dtrsm(bool upper, bool nounit, double* A, double* b) {
	if (upper) {
		#pragma unroll 1
		for (int k = NSP - 1; k >= 0; --k)
		{
			if (nounit) {
				b[k] /= A[k + k * NSP];
			}
			#pragma unroll 1
			for (int i = 0; i < k; i++)
			{
				b[i] -= b[k] * A[i + k * NSP];
			}
		}
	}
	else{
		#pragma unroll 1
		for (int k = 0; k < NSP; k++) {
			if (fabs(b[k]) > 0) {
				if (nounit) {
					b[k] /= A[k + k * NSP];
				}
				#pragma unroll 1
				for (int i = k + 1; i < NSP; i++)
				{
					b[i] -= b[k] * A[i + k * NSP];
				}
			}
		}
	}
}

__device__ void dgetrs(double* A, double* B, int* ipiv) {
	dlaswp(B, ipiv);
	dtrsm(false, false, A, B);
	dtrsm(true, true, A, B);
}

__device__ void zlaswp(hipDoubleComplex* A, int* ipiv) {
	#pragma unroll 1
	for (int i = 0; i < NSP; i++) {
		int ip = ipiv[i];
		if (ip != i) {
			hipDoubleComplex temp = A[i];
			A[i] = A[ip];
			A[ip] = temp;
		}
	}	
}

//diag == 'n' -> nounit = true
//upper == 'u' -> upper = true
__device__ void ztrsm(bool upper, bool nounit, hipDoubleComplex* A, hipDoubleComplex* b) {
	if (upper) {
		#pragma unroll 1
		for (int k = NSP - 1; k >= 0; --k)
		{
			if (nounit) {
				b[k] = hipCdiv(b[k], A[k + k * NSP]);
			}
			#pragma unroll 1
			for (int i = 0; i < k; i++)
			{
				b[i] = hipCsub(b[i], hipCmul(b[k], A[i + k * NSP]));
			}
		}
	}
	else{
		#pragma unroll 1
		for (int k = 0; k < NSP; k++) {
			if (hipCabs(b[k]) > 0) {
				if (nounit) {
					b[k] = hipCdiv(b[k], A[k + k * NSP]);
				}
				#pragma unroll 1
				for (int i = k + 1; i < NSP; i++)
				{
					b[i] = hipCsub(b[i], hipCmul(b[k], A[i + k * NSP]));
				}
			}
		}
	}
}

__device__ void zgetrs(hipDoubleComplex* A, hipDoubleComplex* B, int* ipiv) {
	zlaswp(B, ipiv);
	ztrsm(false, false, A, B);
	ztrsm(true, true, A, B);
}

__device__ void RK_Solve(double H, double* E1, hipDoubleComplex* E2, double* R1, double* R2, double* R3, int* ipiv1, int* ipiv2) {
	// Z = (1/h) T^(-1) A^(-1) * Z
	#pragma unroll 1
	for(int i = 0; i < NSP; i++)
	{
		double x1 = R1[i] / H;
		double x2 = R2[i] / H;
		double x3 = R3[i] / H;
		R1[i] = rkTinvAinv[0][0] * x1 + rkTinvAinv[0][1] * x2 + rkTinvAinv[0][2] * x3;
		R2[i] = rkTinvAinv[1][0] * x1 + rkTinvAinv[1][1] * x2 + rkTinvAinv[1][2] * x3;
		R3[i] = rkTinvAinv[2][0] * x1 + rkTinvAinv[2][1] * x2 + rkTinvAinv[2][2] * x3;
	}
	dgetrs(E1, R1, ipiv1);
	hipDoubleComplex temp[NSP];
	#pragma unroll 1
	for (int i = 0; i < NSP; ++i)
	{
		temp[i] = make_hipDoubleComplex(R2[i], R3[i]);
	}
	zgetrs(E2, temp, ipiv2);
	#pragma unroll 1
	for (int i = 0; i < NSP; ++i)
	{
		R2[i] = hipCreal(temp[i]);
		R3[i] = hipCimag(temp[i]);
	}

	// Z = T * Z
	#pragma unroll 1
	for (int i = 0; i < NSP; ++i) {
		double x1 = R1[i];
		double x2 = R2[i];
		double x3 = R3[i];
		R1[i] = rkT[0][0] * x1 + rkT[0][1] * x2 + rkT[0][2] * x3;
		R2[i] = rkT[1][0] * x1 + rkT[1][1] * x2 + rkT[1][2] * x3;
		R3[i] = rkT[2][0] * x1 + rkT[2][1] * x2 + rkT[2][2] * x3;
	}
}

__device__ double RK_ErrorNorm(double* scale, double* DY) {
	double sum = 0;
	#pragma unroll 1
	for (int i = 0; i < NSP; ++i){
		sum += (scale[i] * scale[i] * DY[i] * DY[i]);
	}
	return fmax(sqrt(sum / ((double)NSP)), 1e-10);
}

__device__ double RK_ErrorEstimate(double H, double t, double pr, double* Y, double* F0, double* Z1, double* Z2, double* Z3, double* scale, double* E1, int* ipiv1, bool FirstStep, bool Reject) {
	double HrkE1  = rkE[1]/H;
    double HrkE2  = rkE[2]/H;
    double HrkE3  = rkE[3]/H;

    double F1[NSP];
    double F2[NSP];
    double TMP[NSP];
    #pragma unroll 1
    for (int i = 0; i < NSP; ++i) {
    	F2[i] = HrkE1 * Z1[i] + HrkE2 * Z2[i] + HrkE3 * Z3[i];
    }
    #pragma unroll 1
    for (int i = 0; i < NSP; ++i) {
    	TMP[i] = rkE[0] * F0[i] + F2[i];
    }
    dgetrs(E1, TMP, ipiv1);
    double Err = RK_ErrorNorm(scale, TMP);
    if (Err >= 1.0 && (FirstStep || Reject)) {
        #pragma unroll 1
    	for (int i = 0; i < NSP; i++) {
        	TMP[i] += Y[i];
        }
    	dydt(t, pr, TMP, F1);
    	#pragma unroll 1
    	for (int i = 0; i < NSP; i++) {
        	TMP[i] = F1[i] + F2[i];
        }
        dgetrs(E1, TMP, ipiv1);
        Err = RK_ErrorNorm(scale, TMP);
    }
    return Err;
}

/** 
 *  5th-order Radau2A implementation
 * 
 */
__device__ void integrate (const double t_start, const double t_end, const double pr, double* y) {
	double Hmin = 0;
	double Hold = 0;
#ifdef Gustafsson
	double Hacc = 0;
	double ErrOld = 0;
#endif
	double H = fmin(5e-7, t_end - t_start);
	double Hnew;
	double t = t_start;
	bool Reject = false;
	bool FirstStep = true;
	bool SkipJac = false;
	bool SkipLU = false;
	double sc[NSP];
	double A[NSP * NSP] = {0.0};
	double E1[NSP * NSP] = {0.0};
	hipDoubleComplex E2[NSP * NSP] = {make_hipDoubleComplex(0.0, 0.0)};
	int ipiv1[NSP] = {0};
	int ipiv2[NSP] = {0};
	double Z1[NSP] = {0.0};
	double Z2[NSP] = {0.0};
	double Z3[NSP] = {0.0};
#ifdef SDIRK_ERROR
	double Z4[NSP] = {0.0};
	double DZ4[NSP] = {0.0};
	double G[NSP] = {0.0};
	double TMP[NSP] = {0.0};
#endif
	double DZ1[NSP] = {0.0};
	double DZ2[NSP] = {0.0};
	double DZ3[NSP] = {0.0};
	double CONT[NSP * 3] = {0.0};
	scale_init(y, sc);
	double y0[NSP];
	safe_memcpy(y0, y);
	double F0[NSP];
	int info = 0;
	int Nconsecutive = 0;
	int Nsteps = 0;
	double NewtonRate = pow(2.0, 1.25);
	while (t + Roundoff < t_end) {
		if(!Reject) {
			dydt (t, pr, y, F0);
		}
		if(!SkipLU) { 
			//need to update Jac/LU
			if(!SkipJac) {
				eval_jacob (t, pr, y, A);
			}
			RK_Decomp(H, E1, E2, A, ipiv1, ipiv2, &info);
			if(info != 0) {
				Nconsecutive += 1;
				if (Nconsecutive >= 5)
				{
					y[0] = logf(-1);
					return;
				}
				H *= 0.5;
				Reject = true;
				SkipJac = true;
				SkipLU = false;
				continue;
			}
			else
			{
				Nconsecutive = 0;
			}
		}
		Nsteps += 1;
		if (Nsteps >= Max_no_steps)
		{
			y[0] = logf(-1);
			return;
		}
		if (0.1 * fabs(H) <= fabs(t) * Roundoff)
		{
			y[0] = logf(-1);
			return;
		}
		if (FirstStep || !StartNewton) {
			safe_memset3(Z1, Z2, Z3, 0);
		} else {
			RK_Interpolate(H, Hold, Z1, Z2, Z3, CONT);
		}
		bool NewtonDone = false;
		double NewtonIncrementOld = 0;
		double Fac = 0.5; //Step reduction if too many iterations
		int NewtonIter = 0;
		double Theta = 0;
		
		//reuse previous NewtonRate
		NewtonRate = pow(fmax(NewtonRate, EPS), 0.8);

		NEWTON_UNROLLER
		for (; NewtonIter < NewtonMaxit; NewtonIter++) {
			RK_PrepareRHS(t, pr, H, y, F0, Z1, Z2, Z3, DZ1, DZ2, DZ3);
			RK_Solve(H, E1, E2, DZ1, DZ2, DZ3, ipiv1, ipiv2);
			double d1 = RK_ErrorNorm(sc, DZ1);
			double d2 = RK_ErrorNorm(sc, DZ2);
			double d3 = RK_ErrorNorm(sc, DZ3);
			double NewtonIncrement = sqrt((d1 * d1 + d2 * d2 + d3 * d3) / 3.0);

			Theta = ThetaMin;
			if (NewtonIter > 0) 
			{
				Theta = NewtonIncrement / NewtonIncrementOld;
				if(Theta >= 0.99) //! Non-convergence of Newton: Theta too large
					break;
				else
					NewtonRate = Theta / (1.0 - Theta);
				//Predict error at the end of Newton process 
				double NewtonPredictedErr = (NewtonIncrement * pow(Theta, (NewtonMaxit - NewtonIter - 1))) / (1.0 - Theta);
				if(NewtonPredictedErr >= NewtonTol) {
					//Non-convergence of Newton: predicted error too large
					double Qnewton = fmin(10.0, NewtonPredictedErr / NewtonTol);
                    Fac = 0.8 * pow(Qnewton, -1.0/((double)(NewtonMaxit-NewtonIter)));
                    break;
				}
			}

			NewtonIncrementOld = fmax(NewtonIncrement, Roundoff);
            // Update solution
            #pragma unroll 1
            for (int i = 0; i < NSP; i++)
            {
            	Z1[i] -= DZ1[i];
            	Z2[i] -= DZ2[i];
            	Z3[i] -= DZ3[i];
            }

            NewtonDone = (NewtonRate * NewtonIncrement <= NewtonTol);
#ifndef NEWTON_UNROLL
            if (NewtonDone) break;
#else //only break if it's at the end of the unroll
            if(NewtonDone && (NewtonIter + 1) % NEWTON_UNROLL == 0) break;
#endif
            if (NewtonIter >= NewtonMaxit)
            {
				y[0] = logf(-1);
				return;
			}
		}
		if(!NewtonDone) {
			H = Fac * H;
			Reject = true;
			SkipJac = true;
			SkipLU = false;
			continue;
		}
#ifdef SDIRK_ERROR
		//!~~~>   Prepare the loop-independent part of the right-hand side
		//!       G = H*rkBgam(0)*F0 + rkTheta(1)*Z1 + rkTheta(2)*Z2 + rkTheta(3)*Z3
		#pragma unroll 1
		for (int i = 0; i < NSP; i++) {
			Z4[i] = Z3[i];
			G[i] = rkBgam[0]*F0[i]*H + rkTheta[0] * Z1[i] + rkTheta[1] * Z2[i] + rkTheta[2] * Z3[i];
		}
		NewtonDone = false;
        Fac = 0.5; // ! Step reduction factor if too many iterations
        double NewtonIncrement = 0;
		
		NEWTON_UNROLLER
        for (int sNewtonIter = 0; sNewtonIter < NewtonMaxit; sNewtonIter++) {
        	//!~~~>   Prepare the loop-dependent part of the right-hand side
        	WADD(y, Z4, TMP);
        	dydt(t + H, pr, TMP, DZ4);
        	#pragma unroll 1
        	for(int i = 0; i < NSP; i++){
        		DZ4[i] += (rkGamma / H) * (G[i] - Z4[i]);
        	}
        	//Solve the linear system
        	dgetrs(E1, DZ4, ipiv1);
        	//Check convergence of Newton iterations
        	NewtonIncrement = RK_ErrorNorm(sc,DZ4);
        	double sNewtonRate = 2.0;
        	double ThetaSD = ThetaMin;
        	if (sNewtonIter > 0) {
            	ThetaSD = NewtonIncrement/NewtonIncrementOld;
            	if (ThetaSD < 0.99) {
            		sNewtonRate = ThetaSD/(1.0-ThetaSD);
                    //! Predict error at the end of Newton process 
                    double NewtonPredictedErr = (NewtonIncrement * pow(ThetaSD, (NewtonMaxit - sNewtonIter - 1))) / (1.0 - Theta);
                    if (NewtonPredictedErr >= NewtonTol) {
                    	//! Non-convergence of Newton: predicted error too large
						double Qnewton = fmin(10.0, NewtonPredictedErr / NewtonTol);
	                    Fac = 0.8 * pow(Qnewton, -1.0/((double)(NewtonMaxit-sNewtonIter)));
	                    break;
                    }
            	}
            	else
            	{
            		//! Non-convergence of Newton: predicted error too large
            		break;
            	}
            }
            NewtonIncrementOld = NewtonIncrement;
            //! Update solution: Z4 <-- Z4 + DZ4
            #pragma unroll 1
            for (int i = 0; i < NSP; i++) {
            	Z4[i] += DZ4[i];
            }

#ifndef NEWTON_UNROLL
            if (NewtonDone) break;
#else //only break if it's at the end of the unroll
            if (NewtonDone && (sNewtonIter + 1) % NEWTON_UNROLL == 0) break;
#endif
        }
        if (!NewtonDone) {
        	H = Fac*H;
        	Reject = true;
        	SkipJac = true;
        	SkipLU = false;
        	continue;
		}
#endif
#ifdef SDIRK_ERROR
		#pragma unroll 1
		for (int i = 0; i < NSP; i++) {
			DZ4[i] = Z3[i] - Z4[i];
		}
		double Err = RK_ErrorNorm(sc, DZ4);
#else
		double Err = RK_ErrorEstimate(H, t, pr, y, F0, Z1, Z2, Z3, sc, E1, ipiv1, FirstStep, Reject);
#endif
		//!~~~> Computation of new step size Hnew
		Fac = pow(Err, (-1.0 / rkELO)) * (1.0 + 2 * NewtonMaxit) / (NewtonIter + 1 + 2 * NewtonMaxit);
		Fac = fmin(FacMax, fmax(FacMin, Fac));
		Hnew = Fac * H;
		if (Err < 1.0) {
#ifdef Gustafsson
			if (!FirstStep) {
				double FacGus = FacSafe * (H / Hacc) * pow(Err * Err / ErrOld, -0.25);
				FacGus = fmin(FacMax, fmax(FacMin, FacGus));
				Fac = fmin(Fac, FacGus);
				Hnew = Fac * H;
			}
			Hacc = H;
			ErrOld = fmax(1e-2, Err);
#endif
			FirstStep = false;
			Hold = H;
			t += H;
			#pragma unroll 1
			for (int i = 0; i < NSP; i++) {
				y[i] += Z3[i];
			}
			// Construct the solution quadratic interpolant Q(c_i) = Z_i, i=1:3
			if (StartNewton) {
				RK_Make_Interpolate(Z1, Z2, Z3, CONT);
			}
			scale(y, y0, sc);
			safe_memcpy(y0, y);
			Hnew = fmin(fmax(Hnew, Hmin), t_end - t);
			if (Reject) {
				Hnew = fmin(Hnew, H);
			}
			Reject = false;
			if (t + Hnew / Qmin - t_end >= 0.0) {
				H = t_end - t;
			} else {
				double Hratio = Hnew / H;
	            // Reuse the LU decomposition
	            SkipLU = (Theta <= ThetaMin) && (Hratio>=Qmin) && (Hratio<=Qmax);
	            if (!SkipLU) H = Hnew;
			}
			// If convergence is fast enough, do not update Jacobian
         	SkipJac = NewtonIter == 1 || NewtonRate <= ThetaMin;
		}
		else {
			if (FirstStep || Reject) {
				H = FacRej * H;
			} else {
				H = Hnew;
			}
			Reject = true;
			SkipJac = true;
			SkipLU = false;
		}
	}
}