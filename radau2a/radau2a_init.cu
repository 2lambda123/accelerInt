#include "hip/hip_runtime.h"
/* radau2a_init.cu
*  Implementation of the necessary initialization for the Radau2A solver
 * \file radau2a_init.cu
 *
 * \author Nicholas Curtis
 * \date 03/09/2015
 *
 */

 #include "header.cuh"

 void init_solver_log() {
 	
 }

 void solver_log() {
 	
 }

void initialize_solver(int padded, solver_memory** h_mem, solver_memory** d_mem) {
  // Allocate storage for the device struct
  cudaErrorCheck( hipMalloc(d_mem, sizeof(solver_memory)) );
  //allocate the device arrays on the host pointer
  cudaErrorCheck( hipMalloc(&((*h_mem)->E1), NSP * NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->E2), NSP * NSP * padded * sizeof(hipDoubleComplex)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->ipiv1), NSP * padded * sizeof(int)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->ipiv2), NSP * padded * sizeof(int)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->sc), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->Z1), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->Z2), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->Z3), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->DZ1), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->DZ2), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->DZ3), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->CONT), 3 * NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->y0), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->F0), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work1), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work2), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work3), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work4), NSP * padded * sizeof(hipDoubleComplex)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->result), padded * sizeof(double)) );

  //copy host struct to device
  cudaErrorCheck( hipMemcpy(*d_mem, *h_mem, sizeof(solver_memory), hipMemcpyHostToDevice) );
}

 const char* solver_name() {
 	const char* name = "radau2a-int-gpu";
 	return name;
 }

 void calculate_required_size() {
 	//return the size (in bytes), needed per cuda thread
 	size_t num_bytes = 0;
 	//Jacobian (doubles as factorized)
 	num_bytes += NSP * NSP;
 	//two pivot index arrays
 	num_bytes += 2 * NSP;
 	//an error scale array
 	num_bytes += NSP;
 	//6 RHS and interpolant arrays
 	num_bytes += 6 * NSP;
 	//continuation array of size 3 * NSP
 	num_bytes += 3 * NSP;
 	//y0
 	num_bytes += NSP;
 	//F0
 	num_bytes += NSP;
 	//3 work arrays
 	num_bytes += 3 * NSP;
  //result flag
  num_bytes += 1;
 	//convert to bytes
 	num_bytes *= sizeof(double);
 	//and add complex jacobian factorization
 	num_bytes += NSP * NSP * sizeof(hipDoubleComplex);
 }

 void cleanup_solver() {
 	//nothing to do
 }

void free_gpu_memory(solver_memory** h_mem, solver_memory** d_mem, double** d_y, double** d_var)
{
  cudaErrorCheck(hipFree((*h_mem)->spec_rates));
  cudaErrorCheck(hipFree((*h_mem)->rev_rates));
  cudaErrorCheck(hipFree((*h_mem)->conc));
  cudaErrorCheck(hipFree((*h_mem)->dy));
  cudaErrorCheck(hipFree((*h_mem)->dBdT));
  cudaErrorCheck(hipFree((*h_mem)->cp));
  cudaErrorCheck(hipFree((*h_mem)->dot_prod));
  cudaErrorCheck(hipFree((*h_mem)->h));
  cudaErrorCheck(hipFree((*h_mem)->fwd_rates));
  cudaErrorCheck(hipFree((*h_mem)->pres_mod));
  cudaErrorCheck(hipFree((*h_mem)->y));
  cudaErrorCheck(hipFree((*h_mem)->jac));
  cudaErrorCheck(hipFree(*d_mem));
  cudaErrorCheck(hipFree(*d_y));
  cudaErrorCheck(hipFree(*d_var));
}