#include "hip/hip_runtime.h"
/* radau2a_init.cu
*  Implementation of the necessary initialization for the Radau2A solver
 * \file radau2a_init.cu
 *
 * \author Nicholas Curtis
 * \date 03/09/2015
 *
 */

 #include "header.cuh"

 void init_solver_log() {
 	
 }

 void solver_log() {
 	
 }

 const char* solver_name() {
 	const char* name = "radau2a-int-gpu";
 	return name;
 }

 void required_solver_size() {
 	//return the size (in bytes), needed per cuda thread
 	size_t num_bytes = 0;
 	//an error scale array
 	num_bytes += NSP;
 	//6 RHS and interpolant arrays
 	num_bytes += 6 * NSP;
 	//continuation array of size 3 * NSP
 	num_bytes += 3 * NSP;
 	//y0
 	num_bytes += NSP;
 	//3 work arrays
 	num_bytes += 3 * NSP;
 	//convert to bytes
 	num_bytes *= sizeof(double);
  //result flag
  num_bytes += sizeof(int);
  //two pivot index arrays
  num_bytes += 2 * NSP * sizeof(int);
 	//and add complex jacobian factorization
 	num_bytes += NSP * NSP * sizeof(hipDoubleComplex);
 }

void initialize_solver(int padded, solver_memory** h_mem, solver_memory** d_mem) {
  // Allocate storage for the device struct
  cudaErrorCheck( hipMalloc(d_mem, sizeof(solver_memory)) );
  //allocate the device arrays on the host pointer
  cudaErrorCheck( hipMalloc(&((*h_mem)->E2), NSP * NSP * padded * sizeof(hipDoubleComplex)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->ipiv1), NSP * padded * sizeof(int)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->ipiv2), NSP * padded * sizeof(int)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->sc), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->Z1), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->Z2), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->Z3), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->DZ1), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->DZ2), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->DZ3), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->CONT), 3 * NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->y0), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work1), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work2), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work3), NSP * padded * sizeof(double)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->work4), NSP * padded * sizeof(hipDoubleComplex)) );
  cudaErrorCheck( hipMalloc(&((*h_mem)->result), padded * sizeof(double)) );

  //copy host struct to device
  cudaErrorCheck( hipMemcpy(*d_mem, *h_mem, sizeof(solver_memory), hipMemcpyHostToDevice) );
}

 void cleanup_solver(solver_memory** h_mem, solver_memory** d_mem) {
  cudaErrorCheck(hipFree((*h_mem)->E2));
  cudaErrorCheck(hipFree((*h_mem)->ipiv1));
  cudaErrorCheck(hipFree((*h_mem)->ipiv2));
  cudaErrorCheck(hipFree((*h_mem)->Z1));
  cudaErrorCheck(hipFree((*h_mem)->Z2));
  cudaErrorCheck(hipFree((*h_mem)->Z3));
  cudaErrorCheck(hipFree((*h_mem)->DZ1));
  cudaErrorCheck(hipFree((*h_mem)->DZ2));
  cudaErrorCheck(hipFree((*h_mem)->DZ3));
  cudaErrorCheck(hipFree((*h_mem)->CONT));
  cudaErrorCheck(hipFree((*h_mem)->y0));
  cudaErrorCheck(hipFree((*h_mem)->work1));
  cudaErrorCheck(hipFree((*h_mem)->work2));
  cudaErrorCheck(hipFree((*h_mem)->work3));
  cudaErrorCheck(hipFree((*h_mem)->work4));
  cudaErrorCheck(hipFree(d_mem));
}