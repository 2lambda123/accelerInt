#include "header.cuh"
#include "solver_props.cuh"
#include <hip/hip_complex.h>
///////////////////////////////////////////////////////////

__device__
int getComplexMax (const int n, const hipDoubleComplex *complexArr) {
	
	int maxInd = 0;
	if (n == 1)
		return maxInd;
	
	double maxVal = hipCabs(complexArr[INDEX(0)]);
	for (int i = 1; i < n; ++i) {
		if (hipCabs(complexArr[INDEX(i)]) > maxVal) {
			maxInd = i;
			maxVal = hipCabs(complexArr[INDEX(i)]);
		}
	}
	
	return maxInd;
}

///////////////////////////////////////////////////////////

__device__
void scaleComplex (const int n, const hipDoubleComplex val, hipDoubleComplex* arrX) {
	
	for (int i = 0; i < n; ++i) {
		arrX[INDEX(i)] = hipCmul(arrX[INDEX(i)], val);
	}
	
}

///////////////////////////////////////////////////////////

__device__
void swapComplex (const int n, hipDoubleComplex* arrX, const int incX, hipDoubleComplex* arrY, const int incY) {
	
	int ix = 0;
	int iy = 0;
	
	for (int i = 0; i < n; ++i) {
		hipDoubleComplex temp = arrX[INDEX(ix)];
		arrX[INDEX(ix)] = arrY[INDEX(iy)];
		arrY[INDEX(iy)] = temp;
		ix += incX;
		iy += incY;
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGERU (const int n, const hipDoubleComplex alpha, const hipDoubleComplex* arrX,
									const hipDoubleComplex* arrY, const int incY, hipDoubleComplex* A, const int lda) {
	
	for (int j = 0; j < n; ++j) {
    	if (hipCabs(arrY[INDEX(j * incY)]) > 0.0) {
      
			hipDoubleComplex temp = hipCmul(alpha, arrY[INDEX(j * incY)]);
      
			for (int i = 0; i < n; ++i) {
				A[INDEX(i + (lda * j))] = hipCfma(arrX[INDEX(i)], temp, A[INDEX(i + (lda * j))]);
			}
      
		}    
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexLU (hipDoubleComplex* A, int* indPivot, int* info) {
	
	for (int j = 0; j < NSP; ++j) {
		
		// find pivot and test for singularity
		
		int jp = j + getComplexMax (NSP - j, &A[INDEX(j + (NSP * j))]);
		indPivot[INDEX(j)] = jp;

		if (hipCabs(A[INDEX(jp + (NSP * j))]) > 0.0) {
			
			// apply interchange to columns 1:n-1
			if (jp != j)
				swapComplex (NSP, &A[INDEX(j)], NSP, &A[INDEX(jp)], NSP);
			
			// compute elements j+1:m-1 of the jth column
			
			if (j < NSP - 1)
				scaleComplex (NSP - j - 1, hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[INDEX(j + (NSP * j))]), &A[INDEX(j + 1 + (NSP * j))]);
			
		} else if (*info == 0) {
			*info = j;
			break;
		}
		
		// update trailing submatrix
		if (j < NSP - 1)
			complexGERU (NSP - j - 1, make_hipDoubleComplex(-1.0, 0.0), &A[INDEX(j + 1 + (NSP * j))], &A[INDEX(j + NSP * (j + 1))], NSP, &A[INDEX(j + 1 + NSP * (j + 1))], NSP);
		
	}
}