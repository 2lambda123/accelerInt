#include "header.cuh"
#include "solver_props.cuh"
#include <hip/hip_complex.h>
///////////////////////////////////////////////////////////

__device__
int getComplexMax (const int n, const hipDoubleComplex *complexArr) {
	
	int maxInd = 0;
	if (n == 1)
		return maxInd;
	
	double maxVal = hipCabs(complexArr[0]);
	for (int i = 1; i < n; ++i) {
		if (hipCabs(complexArr[i]) > maxVal) {
			maxInd = i;
			maxVal = hipCabs(complexArr[i]);
		}
	}
	
	return maxInd;
}

///////////////////////////////////////////////////////////

__device__
void scaleComplex (const int n, const hipDoubleComplex val, hipDoubleComplex* arrX) {
	
	for (int i = 0; i < n; ++i) {
		arrX[i] = hipCmul(arrX[i], val);
	}
	
}

///////////////////////////////////////////////////////////

__device__
void swapComplex (const int n, hipDoubleComplex* arrX, const int incX, hipDoubleComplex* arrY, const int incY) {
	
	int ix = 0;
	int iy = 0;
	
	for (int i = 0; i < n; ++i) {
		hipDoubleComplex temp = arrX[ix];
		arrX[ix] = arrY[iy];
		arrY[iy] = temp;
		ix += incX;
		iy += incY;
	}
	
}

///////////////////////////////////////////////////////////

__device__
void complexGERU (const int n, const hipDoubleComplex alpha, const hipDoubleComplex* arrX,
									const hipDoubleComplex* arrY, const int incY, hipDoubleComplex* A, const int lda) {
	
	for (int j = 0; j < n; ++j) {
    	if (hipCabs(arrY[j * incY]) > 0.0) {
      
			hipDoubleComplex temp = hipCmul(alpha, arrY[j * incY]);
      
			for (int i = 0; i < n; ++i) {
				A[i + (lda * j)] = hipCfma(arrX[i], temp, A[i + (lda * j)]);
			}
      
		}    
	}
	
}

///////////////////////////////////////////////////////////

__device__
void getComplexLU (hipDoubleComplex* A, int* indPivot, int* info) {
	
	for (int j = 0; j < NSP; ++j) {
		
		// find pivot and test for singularity
		
		int jp = j + getComplexMax (NSP - j, &A[j + (NSP * j)]);
		indPivot[j] = jp;

		if (hipCabs(A[jp + (NSP * j)]) > 0.0) {
			
			// apply interchange to columns 1:n-1
			if (jp != j)
				swapComplex (NSP, &A[j], NSP, &A[jp], NSP);
			
			// compute elements j+1:m-1 of the jth column
			
			if (j < NSP - 1)
				scaleComplex (NSP - j - 1, hipCdiv(make_hipDoubleComplex(1.0, 0.0), A[j + (NSP * j)]), &A[j + 1 + (NSP * j)]);
			
		} else if (*info == 0) {
			*info = j;
			break;
		}
		
		// update trailing submatrix
		if (j < NSP - 1)
			complexGERU (NSP - j - 1, make_hipDoubleComplex(-1.0, 0.0), &A[j + 1 + (NSP * j)], &A[j + NSP * (j + 1)], NSP, &A[j + 1 + NSP * (j + 1)], NSP);
		
	}
}